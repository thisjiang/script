#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/math/math_cuda_utils.h"
#include "paddle/fluid/operators/softmax_op.h"
#include "paddle/fluid/platform/cuda_device_function.h"
#ifdef PADDLE_WITH_HIP
#include "paddle/fluid/platform/miopen_helper.h"
#else
#include "paddle/fluid/platform/cudnn_helper.h"
#endif
#include "paddle/fluid/platform/gpu_launch_config.h"

namespace paddle {
namespace platform {
struct CUDAPlace;
struct float16;
}  // namespace platform
}  // namespace paddle

namespace paddle {
namespace operators {

using ScopedTensorDescriptor = platform::ScopedTensorDescriptor;
using DataLayout = platform::DataLayout;
using Tensor = framework::Tensor;

#define LAUNCH_SOFTMAX_WARP_FORWARD(Log2Elements)                  \
  case Log2Elements:                                               \
    WarpSoftmaxForward<T, float, Log2Elements><<<                  \
        blocks, threads, 0, ctx.cuda_device_context().stream()>>>( \
        out_data, x->data<T>(), N, dim, dim);                      \
    break;

#define LAUNCH_SOFTMAX_WARP_BACKWARD(Log2Elements)                 \
  case Log2Elements:                                               \
    softmax_warp_backward<T, float, Log2Elements><<<               \
        blocks, threads, 0, ctx.cuda_device_context().stream()>>>( \
        dx_data, mul_grad.data<T>(), out->data<T>(), N, dim, dim); \
    break;

static inline int SizeOutAxis(const int axis, DDim dims) {
  int size = 1;
  for (int i = axis + 1; i < dims.size(); i++) {
    size *= dims[i];
  }
  return size;
}

int log2_ceil(int value) {
  int log2_value = 0;
  while ((1 << log2_value) < value) ++log2_value;
  return log2_value;
}

template <typename T, int VLEN>
union vec_t {
  static_assert(sizeof(T) == -1, "vec_t is only available by specialization.");
};

template <>
union vec_t<float, 4> {
  float4 s;
  float v[4];
};

template <>
union vec_t<platform::float16, 4> {
  int2 s;
  platform::float16 v[4];
};

template <typename T, typename VECT, int VPT, int WARP_PER_BLOCK>
__global__ void VecSoftmaxForward(T* dst, const T* src, const int batch_size,
                                  const int softmax_ele) {
  int offset = blockIdx.x * softmax_ele * WARP_PER_BLOCK;
  int idx = threadIdx.x * VPT;

  VECT buf = reinterpret_cast<const VECT*>(&src[offset + idx])[0];
  T* bufp = reinterpret_cast<T*>(&buf);
  float4 val4;
  float* val4p = reinterpret_cast<float*>(&val4);
  for (int i = 0; i < VPT; ++i) {
    val4p[i] = static_cast<float>(bufp[i]);
  }
  float val = val4.x + val4.y + val4.z + val4.w;
  float max_val = math::warpReduceMax<float>(
      max(max(val4.x, val4.y), max(val4.z, val4.w)), 0xffffffff);
  float4 tmp4 = make_float4(__expf(val4.x - max_val), __expf(val4.y - max_val),
                            __expf(val4.z - max_val), __expf(val4.w - max_val));
  float* tmp4p = reinterpret_cast<float*>(&tmp4);
  float invsum = 1.f / (math::warpReduceSum<float>(
                            tmp4.x + tmp4.y + tmp4.z + tmp4.w, 0xffffffff) +
                        1e-6f);
  for (int i = 0; i < VPT; ++i) {
    bufp[i] = static_cast<T>(tmp4p[i] * invsum);
  }
  reinterpret_cast<VECT*>(&dst[offset + idx])[0] = buf;
}

template <typename T, int WARP_BATCH, int WARP_SIZE_SOFTMAX>
__device__ __forceinline__ void warp_reduce_sum(T* sum) {
#pragma unroll
  for (int offset = WARP_SIZE_SOFTMAX / 2; offset > 0; offset /= 2) {
#pragma unroll
    for (int i = 0; i < WARP_BATCH; ++i) {
      T sum_val = platform::CudaShuffleXorSync(0xFFFFFFFF, sum[i], offset);
      sum[i] = sum[i] + sum_val;
    }
  }
}

template <typename T, int WARP_BATCH, int WARP_SIZE_SOFTMAX>
__device__ __forceinline__ void warp_reduce_max(T* sum) {
#pragma unroll
  for (int offset = WARP_SIZE_SOFTMAX / 2; offset > 0; offset /= 2) {
#pragma unroll
    for (int i = 0; i < WARP_BATCH; ++i) {
      T max_val = platform::CudaShuffleXorSync(0xFFFFFFFF, sum[i], offset);
      sum[i] = max(sum[i], max_val);
    }
  }
}

template <typename T, typename AccT, int Log2Elements>
__global__ void WarpSoftmaxForward(T* dst, const T* src, const int batch_size,
                                   const int stride, const int element_count) {
  constexpr int next_power_of_two = 1 << Log2Elements;
  constexpr int warp_size_softmax =
      (next_power_of_two < 32) ? next_power_of_two : 32;
  constexpr int WARP_ITERATIONS = next_power_of_two / warp_size_softmax;
  constexpr int WARP_BATCH = (next_power_of_two <= 128) ? 2 : 1;

  int first_batch = (blockDim.y * blockIdx.x + threadIdx.y) * WARP_BATCH;

  int local_batches = batch_size - first_batch;
  if (local_batches > WARP_BATCH) {
    local_batches = WARP_BATCH;
  }

  int local_idx = threadIdx.x;

  src += first_batch * stride + local_idx;
  dst += first_batch * stride + local_idx;

  // load data from global memory
  AccT elements[WARP_BATCH][WARP_ITERATIONS];
  for (int i = 0; i < WARP_BATCH; ++i) {
    int batch_element_count = (i >= local_batches) ? 0 : element_count;
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      int element_index = local_idx + it * warp_size_softmax;
      if (element_index < batch_element_count) {
        elements[i][it] =
            static_cast<float>(src[i * element_count + it * warp_size_softmax]);
      } else {
        elements[i][it] = -std::numeric_limits<AccT>::infinity();
      }
    }
  }

  // compute max_value
  AccT max_value[WARP_BATCH];
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
    max_value[i] = elements[i][0];
#pragma unroll
    for (int it = 1; it < WARP_ITERATIONS; ++it) {
      max_value[i] =
          (max_value[i] > elements[i][it]) ? max_value[i] : elements[i][it];
    }
  }
  warp_reduce_max<AccT, WARP_BATCH, warp_size_softmax>(max_value);

  AccT sum[WARP_BATCH]{0.0f};
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
#pragma unroll
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      elements[i][it] = (std::exp((elements[i][it] - max_value[i])));
      sum[i] += elements[i][it];
    }
  }
  warp_reduce_sum<AccT, WARP_BATCH, warp_size_softmax>(sum);

// store result
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
    if (i >= local_batches) break;
#pragma unroll
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      int element_index = local_idx + it * warp_size_softmax;
      if (element_index < element_count) {
        dst[i * element_count + it * warp_size_softmax] =
            elements[i][it] / sum[i];
      } else {
        break;
      }
    }
  }
}

template <typename T, typename AccT, int Log2Elements>
__global__ void softmax_warp_backward(T* gradInput, const T* grad,
                                      const T* output, int batch_size,
                                      int stride, int element_count) {
  constexpr int next_power_of_two = 1 << Log2Elements;
  constexpr int warp_size_softmax =
      (next_power_of_two < 32) ? next_power_of_two : 32;
  constexpr int WARP_ITERATIONS = next_power_of_two / warp_size_softmax;
  constexpr int WARP_BATCH = (next_power_of_two <= 128) ? 2 : 1;

  int first_batch = (blockDim.y * blockIdx.x + threadIdx.y) * WARP_BATCH;

  int local_batches = batch_size - first_batch;
  if (local_batches > WARP_BATCH) {
    local_batches = WARP_BATCH;
  }

  int local_idx = threadIdx.x % warp_size_softmax;

  int thread_offset = first_batch * stride + local_idx;
  grad += thread_offset;
  output += thread_offset;
  gradInput += thread_offset;

  // load data from global memory
  AccT grad_reg[WARP_BATCH][WARP_ITERATIONS];
  AccT output_reg[WARP_BATCH][WARP_ITERATIONS];
  for (int i = 0; i < WARP_BATCH; ++i) {
    int batch_element_count = (i >= local_batches) ? 0 : element_count;
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      int element_index = local_idx + it * warp_size_softmax;
      if (element_index < batch_element_count) {
        grad_reg[i][it] =
            static_cast<AccT>(grad[i * element_count + it * warp_size_softmax]);
        output_reg[i][it] = static_cast<AccT>(
            output[i * element_count + it * warp_size_softmax]);
      } else {
        grad_reg[i][it] = AccT(0);
        output_reg[i][it] = AccT(0);
      }
    }
  }

  AccT sum[WARP_BATCH];
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
    sum[i] = grad_reg[i][0];
#pragma unroll
    for (int it = 1; it < WARP_ITERATIONS; ++it) {
      sum[i] += grad_reg[i][it];
    }
  }
  warp_reduce_sum<AccT, WARP_BATCH, warp_size_softmax>(sum);

// store result
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
    if (i >= local_batches) break;
#pragma unroll
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      int element_index = local_idx + it * warp_size_softmax;
      if (element_index < element_count) {
        // compute gradients
        gradInput[i * element_count + it * warp_size_softmax] =
            (grad_reg[i][it] - output_reg[i][it] * sum[i]);
      }
    }
  }
}

template <typename T>
__global__ void MultiplyCUDAKernel(T* C, const T* A, const T* B, int N) {
  CUDA_KERNEL_LOOP(i, N) {
    C[i] = static_cast<T>(static_cast<float>(A[i]) * static_cast<float>(B[i]));
  }
}

template <typename T, int VPT, int WARP_PER_BLOCK>
__global__ void VecSoftmaxBackward(T* dst, const T* grad, const T* src,
                                   const int batch_size,
                                   const int softmax_ele) {
  const int offset =
      blockIdx.x * softmax_ele * WARP_PER_BLOCK + threadIdx.x * VPT;

  float local_sum_gy = 0.f;
  vec_t<T, VPT> local_grad;
  vec_t<T, VPT> local_src;

  local_grad.s =
      reinterpret_cast<const decltype(local_grad.s)*>(&grad[offset])[0];
  local_src.s = reinterpret_cast<const decltype(local_src.s)*>(&src[offset])[0];

  for (int i = 0; i < VPT; ++i) {
    local_sum_gy += static_cast<float>(local_grad.v[i]) *
                    static_cast<float>(local_src.v[i]);
  }
  float sum_gy = math::warpReduceSum<float>(local_sum_gy, 0xffffffff);

  vec_t<T, VPT> local_dst;
  for (int i = 0; i < VPT; ++i) {
    local_dst.v[i] =
        static_cast<T>(static_cast<float>(local_src.v[i]) *
                       (static_cast<float>(local_grad.v[i]) - sum_gy));
  }
  reinterpret_cast<decltype(local_dst.s)*>(&dst[offset])[0] = local_dst.s;
}

template<typename T>
__forceinline__ __device__ T Exp(const T val) {
  return exp(val);
}
template<>
__forceinline__ __device__ float Exp<float>(const float val) {
  return __expf(val);
}
// When D is larg and dim is small:
// Each block arranged by N，each thread arranged by D
// each thread compute dim number's softmax
template<typename T, typename AccT>
__global__ void KeLoopDimSoftmaxForward(T *dst, const T *src, const int N,
                                      const int dim, const int D) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int out_id = tid / D;
  if(out_id >= N) return;
  const int in_id = tid - out_id * D;

  const T *out_src = src + out_id * dim * D + in_id;
  T *out_dst = dst + out_id * dim * D + in_id;
  // compute max value
  AccT max_val = -std::numeric_limits<AccT>::infinity();
  for(int dim_id = 0; dim_id < dim; dim_id ++)
    max_val = max(static_cast<AccT>(out_src[dim_id * D]), max_val);

  // compute exponent value and sum value
  AccT sum_val(0);
  for(int dim_id = 0; dim_id < dim; dim_id ++)
    sum_val += Exp(static_cast<AccT>(out_src[dim_id * D]) - max_val);

  // compute softmax value
  for(int dim_id = 0; dim_id < dim; dim_id ++)
    out_dst[dim_id * D] =
        static_cast<T>(Exp(static_cast<AccT>(out_src[dim_id * D]) - max_val) /
        (sum_val + 1e-6f));
}

// When D is small and (dim * D) is larger
// Each block arranged by N，each thread arranged by dim * D
// each block compute (dim * D) number's softmax
template<typename T, typename AccT>
__global__ void KeSpandDimDSoftmaxForward(T *dst, const T *src, const int N,
                                      const int dim, const int D) {
  extern __shared__ char s_mem[];
  AccT* s_data = reinterpret_cast<AccT*>(s_mem);

  const int tid = threadIdx.x;
  const int BlockDim = blockDim.x;
  for(int out_id = blockIdx.x; out_id < N; out_id += gridDim.x) {
    const T *out_src = src + out_id * dim * D;
    T *out_dst = dst + out_id * dim * D;

    // Compute each thread's max value
    AccT max_val = -std::numeric_limits<AccT>::infinity();
    for(int id = tid; id < dim * D; id += BlockDim)
      max_val = max(static_cast<AccT>(out_src[id]), max_val);
    // write to shared memory
    s_data[tid] = max_val;
    __syncthreads();
    // compute total max value
    for(int id = tid % D; id < BlockDim; id += D)
      max_val = max(s_data[id], max_val);

    // Compute each thread's sum value
    AccT sum_val(0);
    for(int id = tid; id < dim * D; id += BlockDim)
      sum_val += Exp(static_cast<AccT>(out_src[id]) - max_val);
    // write to shared memory
    s_data[tid] = sum_val;
    __syncthreads();
    // compute total sum value
    sum_val = 0;
    for(int id = tid % D; id < BlockDim; id += D)
      sum_val += s_data[id];

    // Compute finally softmax result
    for(int id = tid; id < dim * D; id += BlockDim)
      out_dst[id] = static_cast<T>(Exp(static_cast<AccT>(out_src[id]) - max_val) /
                      (sum_val + 1e-6f));
  }
}

// When D is small and (dim * D) is larger
// Each block arranged by N，each thread arranged by dim * D
// each block compute (dim * D) number's softmax
template<typename T, typename AccT>
__global__ void KeSpandDimDSoftmaxBackward(T *dx, const T *out, const T *dout,
                        const int N, const int dim, const int D) {
  extern __shared__ char s_mem[];
  AccT* s_data = reinterpret_cast<AccT*>(s_mem);

  const int tid = threadIdx.x;
  const int BlockDim = blockDim.x;
  for(int out_id = blockIdx.x; out_id < N; out_id += gridDim.x) {
    const T *src_out = out + out_id * dim * D;
    const T *src_dout = dout + out_id * dim * D;
    T *dst_dx = dx + out_id * dim * D;

    // Compute each thread's sum value
    AccT sum_val(0);
    for(int id = tid; id < dim * D; id += BlockDim)
      sum_val += static_cast<AccT>(src_out[id]) *
                 static_cast<AccT>(src_dout[id]);
    // write to shared memory
    s_data[tid] = sum_val;
    __syncthreads();
    // compute total sum value
    sum_val = 0;
    for(int id = tid % D; id < BlockDim; id += D)
      sum_val += s_data[id];

    // Compute finally softmax result
    for(int id = tid; id < dim * D; id += BlockDim)
      dst_dx[id] =
        static_cast<T>(static_cast<AccT>(src_out[id]) *
        (static_cast<AccT>(src_dout[id]) - sum_val));
  }
}

template <typename T>
class SoftmaxCUDNNKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* x = ctx.Input<Tensor>("X");
    auto* out = ctx.Output<Tensor>("Out");
    out->mutable_data<T>(ctx.GetPlace());
    auto* out_data = out->data<T>();

    auto dims = x->dims();
    const int rank = dims.size();
    const int axis = CanonicalAxis(ctx.Attr<int>("axis"), rank);
    const int dim = dims[axis];
    const int N = SizeToAxis(axis, dims);
    const int D = SizeOutAxis(axis, dims);

    constexpr int max_dim = 320;
    bool optimize = false;
    constexpr bool AccT_use_float =
        std::is_same<T, float>::value ||
        std::is_same<T, platform::float16>::value;
    if(D <= 512 && D < dim) {
      optimize = true;
      const int grids = N;
      const int threads = D * (512 / D);

      if(AccT_use_float) {
        KeSpandDimDSoftmaxForward<T, float>
          <<<grids, threads, threads * sizeof(float),
            ctx.cuda_device_context().stream()>>>(
          out_data, x->data<T>(), N, dim, D);
      } else {
        KeSpandDimDSoftmaxForward<T, double>
          <<<grids, threads, threads * sizeof(double),
            ctx.cuda_device_context().stream()>>>(
          out_data, x->data<T>(), N, dim, D);
      }
    } else if(dim < 1024) {
      optimize = true;
      int threads = std::min(N * D, 256);
      int grids = (N * D + threads - 1) / threads;

      if(AccT_use_float) {
        KeLoopDimSoftmaxForward<T, float><<<grids, threads, 0,
          ctx.cuda_device_context().stream()>>>(
          out_data, x->data<T>(), N, dim, D);
      } else {
        KeLoopDimSoftmaxForward<T, double><<<grids, threads, 0,
          ctx.cuda_device_context().stream()>>>(
          out_data, x->data<T>(), N, dim, D);
      }
    }
    if (!optimize) {
      ScopedTensorDescriptor desc;
      std::vector<int> tensor_dims = {N, dim, D, 1};
      DataLayout layout = DataLayout::kNCHW;
#ifdef PADDLE_WITH_HIP
      miopenTensorDescriptor_t desc_ = desc.descriptor<T>(layout, tensor_dims);
#else
      hipdnnTensorDescriptor_t desc_ = desc.descriptor<T>(layout, tensor_dims);
#endif

      auto& dev_ctx =
          ctx.template device_context<platform::CUDADeviceContext>();
      auto handle = dev_ctx.cudnn_handle();

#ifdef PADDLE_WITH_HIP
      auto mode = axis == rank - 1 ? MIOPEN_SOFTMAX_MODE_INSTANCE
                                   : MIOPEN_SOFTMAX_MODE_CHANNEL;
      PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::miopenSoftmaxForward(
          handle, platform::CudnnDataType<T>::kOne(), desc_, x->data<T>(),
          platform::CudnnDataType<T>::kZero(), desc_, out_data));
#else
      auto mode = axis == rank - 1 ? HIPDNN_SOFTMAX_MODE_INSTANCE
                                   : HIPDNN_SOFTMAX_MODE_CHANNEL;
      PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSoftmaxForward(
          handle, HIPDNN_SOFTMAX_ACCURATE, mode,
          platform::CudnnDataType<T>::kOne(), desc_, x->data<T>(),
          platform::CudnnDataType<T>::kZero(), desc_, out_data));
#endif
    }
  }
};

template <typename T>
class SoftmaxGradCUDNNKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* out = ctx.Input<Tensor>("Out");
    auto* dout = ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto* dx = ctx.Output<Tensor>(framework::GradVarName("X"));
    dx->mutable_data<T>(ctx.GetPlace());
    auto* dx_data = dx->data<T>();

    auto dims = out->dims();
    const int rank = dims.size();
    const int axis = CanonicalAxis(ctx.Attr<int>("axis"), rank);
    const int dim = dims[axis];
    const int N = SizeToAxis(axis, dims);
    const int D = SizeOutAxis(axis, dims);

    constexpr bool AccT_use_float =
        std::is_same<T, float>::value ||
        std::is_same<T, platform::float16>::value;
    bool optimize = false;
    if(D <= 1024) {
      optimize = true;
      const int grids = N;
      const int threads = D * (1024 / D);

      if(AccT_use_float) {
        KeSpandDimDSoftmaxBackward<T, float>
          <<<grids, threads, threads * sizeof(float),
          ctx.cuda_device_context().stream()>>>(
          dx_data, out->data<T>(), dout->data<T>(), N, dim, D);
      } else {
        KeSpandDimDSoftmaxBackward<T, double>
          <<<grids, threads, threads * sizeof(double),
          ctx.cuda_device_context().stream()>>>(
          dx_data, out->data<T>(), dout->data<T>(), N, dim, D);
      }
    }
    if (!optimize) {
      ScopedTensorDescriptor desc;
      std::vector<int> tensor_dims = {N, dim, D, 1};
      DataLayout layout = DataLayout::kNCHW;
#ifdef PADDLE_WITH_HIP
      miopenTensorDescriptor_t desc_ = desc.descriptor<T>(layout, tensor_dims);
#else
      hipdnnTensorDescriptor_t desc_ = desc.descriptor<T>(layout, tensor_dims);
#endif

      auto& dev_ctx =
          ctx.template device_context<platform::CUDADeviceContext>();
      auto handle = dev_ctx.cudnn_handle();

#ifdef PADDLE_WITH_HIP
      auto mode = axis == rank - 1 ? MIOPEN_SOFTMAX_MODE_INSTANCE
                                   : MIOPEN_SOFTMAX_MODE_CHANNEL;
      PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::miopenSoftmaxBackward(
          handle, platform::CudnnDataType<T>::kOne(), desc_, out->data<T>(),
          desc_, dout->data<T>(), platform::CudnnDataType<T>::kZero(), desc_,
          dx_data));
#else
      auto mode = axis == rank - 1 ? HIPDNN_SOFTMAX_MODE_INSTANCE
                                   : HIPDNN_SOFTMAX_MODE_CHANNEL;
      PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSoftmaxBackward(
          handle, HIPDNN_SOFTMAX_ACCURATE, mode,
          platform::CudnnDataType<T>::kOne(), desc_, out->data<T>(), desc_,
          dout->data<T>(), platform::CudnnDataType<T>::kZero(), desc_,
          dx_data));
#endif
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
REGISTER_OP_KERNEL(softmax, CUDNN, plat::CUDAPlace,
                   ops::SoftmaxCUDNNKernel<float>,
                   ops::SoftmaxCUDNNKernel<plat::float16>);
REGISTER_OP_KERNEL(softmax_grad, CUDNN, plat::CUDAPlace,
                   ops::SoftmaxGradCUDNNKernel<float>,
                   ops::SoftmaxGradCUDNNKernel<plat::float16>);
#else
REGISTER_OP_KERNEL(softmax, CUDNN, plat::CUDAPlace,
                   ops::SoftmaxCUDNNKernel<float>,
                   ops::SoftmaxCUDNNKernel<double>,
                   ops::SoftmaxCUDNNKernel<plat::float16>);
REGISTER_OP_KERNEL(softmax_grad, CUDNN, plat::CUDAPlace,
                   ops::SoftmaxGradCUDNNKernel<float>,
                   ops::SoftmaxGradCUDNNKernel<double>,
                   ops::SoftmaxGradCUDNNKernel<plat::float16>);
#endif
