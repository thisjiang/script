#include "hip/hip_runtime.h"
/* Copyright (c) 2018 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/math/math_cuda_utils.h"
#include "paddle/fluid/operators/softmax_op.h"
#include "paddle/fluid/platform/cuda_device_function.h"
#ifdef PADDLE_WITH_HIP
#include "paddle/fluid/platform/miopen_helper.h"
#else
#include "paddle/fluid/platform/cudnn_helper.h"
#endif
#include "paddle/fluid/platform/gpu_launch_config.h"

namespace paddle {
namespace platform {
struct CUDAPlace;
struct float16;
}  // namespace platform
}  // namespace paddle

namespace paddle {
namespace operators {

using ScopedTensorDescriptor = platform::ScopedTensorDescriptor;
using DataLayout = platform::DataLayout;
using Tensor = framework::Tensor;

#define LAUNCH_SOFTMAX_WARP_BACKWARD(Log2Elements)                 \
  case Log2Elements:                                               \
    softmax_warp_backward<T, float, Log2Elements><<<               \
        blocks, threads, 0, ctx.cuda_device_context().stream()>>>( \
        dx_data, mul_grad.data<T>(), out->data<T>(), N, dim, dim); \
    break;

static inline int SizeOutAxis(const int axis, DDim dims) {
  int size = 1;
  for (int i = axis + 1; i < dims.size(); i++) {
    size *= dims[i];
  }
  return size;
}

int log2_ceil(int value) {
  int log2_value = 0;
  while ((1 << log2_value) < value) ++log2_value;
  return log2_value;
}

template <typename T, int VLEN>
union vec_t {
  static_assert(sizeof(T) == -1, "vec_t is only available by specialization.");
};

template <>
union vec_t<float, 4> {
  float4 s;
  float v[4];
};

template <>
union vec_t<platform::float16, 4> {
  int2 s;
  platform::float16 v[4];
};

template<typename T> struct GetAccType {using type = T;};
template<> struct GetAccType<paddle::platform::float16> {using type = float;};

template<typename T, int N> struct GetVecType;
template<typename T> struct GetVecType<T, 1> {using type = T;};
template<> struct GetVecType<paddle::platform::float16, 2> {using type = half2;};
template<> struct GetVecType<paddle::platform::float16, 4> {using type = float2;};
template<> struct GetVecType<float, 2> {using type = float2;};
template<> struct GetVecType<float, 4> {using type = float4;};
template<> struct GetVecType<double, 2> {using type = double2;};
template<> struct GetVecType<double, 4> {using type = double4;};

template<typename T>
__forceinline__ __device__ T Exp(const T val) {return exp(val);}
template<>
__forceinline__ __device__ float Exp<float>(const float val) {return __expf(val);}

/*****************************************************************/
// when D == 1 && dim < 320, using WarpSoftmaxForward faster
template <typename T, int WARP_BATCH, int WARP_SIZE_SOFTMAX>
__device__ __forceinline__ void warp_reduce_sum(T* sum) {
#pragma unroll
  for (int offset = WARP_SIZE_SOFTMAX / 2; offset > 0; offset /= 2) {
#pragma unroll
    for (int i = 0; i < WARP_BATCH; ++i) {
      T sum_val = platform::CudaShuffleXorSync(0xFFFFFFFF, sum[i], offset);
      sum[i] = sum[i] + sum_val;
    }
  }
}

template <typename T, int WARP_BATCH, int WARP_SIZE_SOFTMAX>
__device__ __forceinline__ void warp_reduce_max(T* sum) {
#pragma unroll
  for (int offset = WARP_SIZE_SOFTMAX / 2; offset > 0; offset /= 2) {
#pragma unroll
    for (int i = 0; i < WARP_BATCH; ++i) {
      T max_val = platform::CudaShuffleXorSync(0xFFFFFFFF, sum[i], offset);
      sum[i] = max(sum[i], max_val);
    }
  }
}

template <typename T, typename AccT, int Log2Elements>
__global__ void WarpSoftmaxForward(T* dst, const T* src, const int batch_size,
                                   const int stride, const int element_count) {
  constexpr int next_power_of_two = 1 << Log2Elements;
  constexpr int warp_size_softmax =
      (next_power_of_two < 32) ? next_power_of_two : 32;
  constexpr int WARP_ITERATIONS = next_power_of_two / warp_size_softmax;
  constexpr int WARP_BATCH = (next_power_of_two <= 128) ? 2 : 1;

  int first_batch = (blockDim.y * blockIdx.x + threadIdx.y) * WARP_BATCH;

  int local_batches = batch_size - first_batch;
  if (local_batches > WARP_BATCH) {
    local_batches = WARP_BATCH;
  }

  int local_idx = threadIdx.x;

  src += first_batch * stride + local_idx;
  dst += first_batch * stride + local_idx;

  // load data from global memory
  AccT elements[WARP_BATCH][WARP_ITERATIONS];
  for (int i = 0; i < WARP_BATCH; ++i) {
    int batch_element_count = (i >= local_batches) ? 0 : element_count;
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      int element_index = local_idx + it * warp_size_softmax;
      if (element_index < batch_element_count) {
        elements[i][it] =
            static_cast<float>(src[i * element_count + it * warp_size_softmax]);
      } else {
        elements[i][it] = -std::numeric_limits<AccT>::infinity();
      }
    }
  }

  // compute max_value
  AccT max_value[WARP_BATCH];
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
    max_value[i] = elements[i][0];
#pragma unroll
    for (int it = 1; it < WARP_ITERATIONS; ++it) {
      max_value[i] =
          (max_value[i] > elements[i][it]) ? max_value[i] : elements[i][it];
    }
  }
  warp_reduce_max<AccT, WARP_BATCH, warp_size_softmax>(max_value);

  AccT sum[WARP_BATCH]{0.0f};
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
#pragma unroll
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      elements[i][it] = (std::exp((elements[i][it] - max_value[i])));
      sum[i] += elements[i][it];
    }
  }
  warp_reduce_sum<AccT, WARP_BATCH, warp_size_softmax>(sum);

// store result
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
    if (i >= local_batches) break;
#pragma unroll
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      int element_index = local_idx + it * warp_size_softmax;
      if (element_index < element_count) {
        dst[i * element_count + it * warp_size_softmax] =
            elements[i][it] / sum[i];
      } else {
        break;
      }
    }
  }
}

template<typename T>
void LaunchWarpSoftmaxForward(hipStream_t &stream, const T* in_data, T* out_data,
                        const int N, const int dim) {
  int log2_elements = static_cast<int>(log2_ceil(dim));
  const int next_power_of_two = 1 << log2_elements;
  int warp_size = (next_power_of_two < 32) ? next_power_of_two : 32;
  int batches_per_warp = (next_power_of_two <= 128) ? 2 : 1;

  // use 128 threads per block to maximimize gpu utilization
  constexpr int threads_per_block = 128;
  int warps_per_block = (threads_per_block / warp_size);
  int batches_per_block = warps_per_block * batches_per_warp;
  int blocks = (N + batches_per_block - 1) / batches_per_block;
  dim3 threads(warp_size, warps_per_block, 1);

#define LAUNCH_SOFTMAX_WARP_FORWARD(Log2Elements)                  \
  case Log2Elements:                                               \
    WarpSoftmaxForward<T, float, Log2Elements><<<                  \
        blocks, threads, 0, stream>>>(                             \
        out_data, in_data, N, dim, dim);                      \
    break;

  switch (log2_elements) {
    LAUNCH_SOFTMAX_WARP_FORWARD(0);  // 1
    LAUNCH_SOFTMAX_WARP_FORWARD(1);  // 2
    LAUNCH_SOFTMAX_WARP_FORWARD(2);  // 4
    LAUNCH_SOFTMAX_WARP_FORWARD(3);  // 8
    LAUNCH_SOFTMAX_WARP_FORWARD(4);  // 16
    LAUNCH_SOFTMAX_WARP_FORWARD(5);  // 32
    LAUNCH_SOFTMAX_WARP_FORWARD(6);  // 64
    LAUNCH_SOFTMAX_WARP_FORWARD(7);  // 128
    LAUNCH_SOFTMAX_WARP_FORWARD(8);  // 256
    LAUNCH_SOFTMAX_WARP_FORWARD(9);  // 512
    default:
      break;
  }
#undef LAUNCH_SOFTMAX_WARP_FORWARD
}

/*****************************************************************/
// when D == 1 && 320 <= dim <= 1024, using KeD1WarpSoftmaxForward faster,
// each warp compute one row's element,
// each thread compute COLS element of dim and store in register
template<typename T, typename AccT, int COLS, int VECSIZE>
__global__ void KeD1WarpSoftmaxForward(T* __restrict__ dst,
            const T* __restrict__ src,const int N, const int dim) {
  static_assert(COLS % VECSIZE == 0);
  constexpr int num_vec = COLS / VECSIZE;
  const int warp_id = blockIdx.x * blockDim.y + threadIdx.y;
  const int tid = threadIdx.x;

  for(int row = warp_id; row < N; row += gridDim.x * blockDim.y) {
    const int offset = row * dim;
    const T* __restrict__ src_row = src + offset;
    T* __restrict__ dst_row = dst + offset;

    using VecT = typename GetVecType<T, VECSIZE>::type;
    VecT vec; // vectorization for global memory coalescing 
    // Load src data from global memory to register,
    // and compute max value
    AccT buf[COLS]{0};
    AccT max_val = -std::numeric_limits<AccT>::infinity();
    int real_cols = 0;
  #pragma unroll
    for(int col = 0; col < num_vec; col ++) {
      int src_col =  (tid + col * WARP_SIZE) * VECSIZE;
      if(src_col >= dim) break;
      vec = reinterpret_cast<const VecT*>(&src_row[src_col])[0];
      T* buf_src = reinterpret_cast<T*>(&vec);

      AccT* buf_acc = buf + real_cols;
  #pragma unroll
      for(int i = 0; i < VECSIZE; i ++) {
        buf_acc[i] = static_cast<AccT>(buf_src[i]);
        max_val = max(buf_acc[i], max_val);
      }
      real_cols += VECSIZE;
    }
    max_val = math::warpReduceMax(max_val, 0xffffffff);
    // compute sum value
    AccT sum_val(0);
  #pragma unroll
    for(int i = 0; i < COLS; i ++) {
      // "break" set in "for loop" aims to avoid local memory
      if(i >= real_cols) break;
      buf[i] = Exp(buf[i] - max_val);
      sum_val += buf[i];
    }
    sum_val = math::warpReduceSum(sum_val, 0xffffffff);
    // compute softmax result
  #pragma unroll
    for(int col = 0; col < num_vec; col ++) {
      int dst_col =  (tid + col * WARP_SIZE) * VECSIZE;
      if(dst_col >= dim) break;
      T *buf_dst = reinterpret_cast<T*>(&vec);
      AccT* buf_acc = buf + col * VECSIZE;
  #pragma unroll
      for(int i = 0; i < VECSIZE; i ++) {
        buf_dst[i] = static_cast<T>(buf_acc[i] / sum_val);
      }
      reinterpret_cast<VecT*>(&dst_row[dst_col])[0] = vec;
    }
  }
}

template<typename T, int COLS, int VECSIZE>
inline void LaunchD1WarpSoftmaxForwardKernel(hipStream_t &stream,
          const T* in_data, T* out_data, const int N, const int dim) {
  int N_b = std::min(8, N);
  dim3 threads(WARP_SIZE, N_b);
  int grids = (N + N_b - 1) / N_b;
  using AccT = typename GetAccType<T>::type;

  KeD1WarpSoftmaxForward<T, AccT, COLS, VECSIZE>
    <<<grids, threads, 0, stream>>>(
      out_data, in_data, N, dim);
}

#define LAUNCH_D1WARP_COLS(COLS)                          \
  case COLS:                                              \
    LaunchD1WarpSoftmaxForwardKernel<T, COLS, VECSIZE>(   \
            stream, in_data, out_data, N, dim);           \
    break;

template<typename T, int VECSIZE>
typename std::enable_if<VECSIZE == 1, void>::type DispatchD1WarpSoftmaxForward(
                        hipStream_t &stream, const T* in_data, T* out_data,
                        const int N, const int dim, const int cols_per_thread) {
  switch (cols_per_thread) {
    LAUNCH_D1WARP_COLS(1)
    LAUNCH_D1WARP_COLS(2)
    LAUNCH_D1WARP_COLS(3)
    LAUNCH_D1WARP_COLS(4)
    LAUNCH_D1WARP_COLS(5)
    LAUNCH_D1WARP_COLS(6)
    LAUNCH_D1WARP_COLS(7)
    LAUNCH_D1WARP_COLS(8)
    LAUNCH_D1WARP_COLS(9)
    LAUNCH_D1WARP_COLS(10)
    LAUNCH_D1WARP_COLS(11)
    LAUNCH_D1WARP_COLS(12)
    LAUNCH_D1WARP_COLS(13)
    LAUNCH_D1WARP_COLS(14)
    LAUNCH_D1WARP_COLS(15)
    LAUNCH_D1WARP_COLS(16)
    LAUNCH_D1WARP_COLS(17)
    LAUNCH_D1WARP_COLS(18)
    LAUNCH_D1WARP_COLS(19)
    LAUNCH_D1WARP_COLS(20)
    LAUNCH_D1WARP_COLS(21)
    LAUNCH_D1WARP_COLS(22)
    LAUNCH_D1WARP_COLS(23)
    LAUNCH_D1WARP_COLS(24)
    LAUNCH_D1WARP_COLS(25)
    LAUNCH_D1WARP_COLS(26)
    LAUNCH_D1WARP_COLS(27)
    LAUNCH_D1WARP_COLS(28)
    LAUNCH_D1WARP_COLS(29)
    LAUNCH_D1WARP_COLS(30)
    LAUNCH_D1WARP_COLS(31)
    LAUNCH_D1WARP_COLS(32)
    default:
      break;
  }
}

template<typename T, int VECSIZE>
typename std::enable_if<VECSIZE == 2, void>::type DispatchD1WarpSoftmaxForward(
                        hipStream_t &stream, const T* in_data, T* out_data,
                        const int N, const int dim, const int cols_per_thread) {
  switch (cols_per_thread) {
    LAUNCH_D1WARP_COLS(2)
    LAUNCH_D1WARP_COLS(4)
    LAUNCH_D1WARP_COLS(6)
    LAUNCH_D1WARP_COLS(8)
    LAUNCH_D1WARP_COLS(10)
    LAUNCH_D1WARP_COLS(12)
    LAUNCH_D1WARP_COLS(14)
    LAUNCH_D1WARP_COLS(16)
    LAUNCH_D1WARP_COLS(18)
    LAUNCH_D1WARP_COLS(20)
    LAUNCH_D1WARP_COLS(22)
    LAUNCH_D1WARP_COLS(24)
    LAUNCH_D1WARP_COLS(26)
    LAUNCH_D1WARP_COLS(28)
    LAUNCH_D1WARP_COLS(30)
    LAUNCH_D1WARP_COLS(32)
    default:
      break;
  }
}

template<typename T, int VECSIZE>
typename std::enable_if<VECSIZE == 4, void>::type DispatchD1WarpSoftmaxForward(
                        hipStream_t &stream, const T* in_data, T* out_data,
                        const int N, const int dim, const int cols_per_thread) {
  switch (cols_per_thread) {
    LAUNCH_D1WARP_COLS(4)
    LAUNCH_D1WARP_COLS(8)
    LAUNCH_D1WARP_COLS(12)
    LAUNCH_D1WARP_COLS(16)
    LAUNCH_D1WARP_COLS(20)
    LAUNCH_D1WARP_COLS(24)
    LAUNCH_D1WARP_COLS(28)
    LAUNCH_D1WARP_COLS(32)
    default:
      break;
  }
}
#undef LAUNCH_D1WARP_COLS

template<typename T>
inline void LaunchD1WarpSoftmaxForward(hipStream_t &stream, const T* in_data,
                      T* out_data, const int N, const int dim) {
  const int cols_per_thread = (dim + WARP_SIZE - 1) / WARP_SIZE;

  if(dim % 4 == 0 && cols_per_thread % 4 == 0) {
    DispatchD1WarpSoftmaxForward<T, 4>(
      stream, in_data, out_data, N, dim, cols_per_thread);
  } else if(dim % 2 == 0 && cols_per_thread % 2 == 0) {
    DispatchD1WarpSoftmaxForward<T, 2>(
      stream, in_data, out_data, N, dim, cols_per_thread);
  } else {
    DispatchD1WarpSoftmaxForward<T, 1>(
      stream, in_data, out_data, N, dim, cols_per_thread);
  }
}

/*****************************************************************/
// when D == 1 && 1024 < dim <= 4096, using KeD1BlockSharedSoftmaxForward,
// each block compute a row, and synchronization by blockReduce,
// each thread compute VECSIZE elements of dim, and store in shared memory
template<typename T, typename AccT, int VECSIZE>
__global__ void KeD1BlockSharedSoftmaxForward(T* __restrict__ dst,
            const T* __restrict__ src,const int N, const int dim) {
  extern __shared__ __align__(sizeof(AccT)) unsigned char s_mem[];
  AccT* s_data = reinterpret_cast<AccT*>(s_mem);

  const int tid = threadIdx.x;
  // vectorization for global memory coalescing
  using VecT = typename GetVecType<T, VECSIZE>::type;
  VecT vec;
  T* buf_src = reinterpret_cast<T*>(&vec);

  for(int row = blockIdx.x; row < N; row += gridDim.x) {
    const int offset = row * dim;
    const T* __restrict__ src_row = src + offset;
    T* __restrict__ dst_row = dst + offset;

    // compute max value
    AccT max_val = -std::numeric_limits<AccT>::infinity();
    for(int col = tid * VECSIZE; col < dim; col += blockDim.x * VECSIZE) {
      vec = reinterpret_cast<const VecT*>(&src_row[col])[0];
      AccT* buf_s = s_data + col;
#pragma unroll
      for(int i = 0; i < VECSIZE; i ++) {
        buf_s[i] = static_cast<AccT>(buf_src[i]);
        max_val = max(buf_s[i], max_val);
      }
    }
    max_val = math::blockReduceMax(max_val, 0xffffffff);
    // compute sum value
    AccT sum_val(0);
    for(int col = tid; col < dim; col += blockDim.x) {
      AccT tmp_val = Exp(s_data[col] - max_val);
      s_data[col] = tmp_val;
      sum_val += tmp_val;
    }
    sum_val = math::blockReduceSum(sum_val, 0xffffffff);
    // compute softmax result
    for(int col = tid * VECSIZE; col < dim; col += blockDim.x * VECSIZE) {
      T* buf_dst = reinterpret_cast<T*>(&vec);
      AccT* buf_s = s_data + col;
  #pragma unroll
      for(int i = 0; i < VECSIZE; i ++) {
        buf_dst[i] = static_cast<T>(buf_s[i] / sum_val);
      }
      reinterpret_cast<VecT*>(&dst_row[col])[0] = vec;
    }
  }
}

template<typename T, int VECSIZE>
inline void LaunchD1BlockSharedSoftmaxForwardKernel(hipStream_t &stream,
                const T* in_data, T* out_data, const int N, const int dim) {
  const int threads = std::min(dim, 256);
  const int grids = N;
  using AccT = typename GetAccType<T>::type;

  KeD1BlockSharedSoftmaxForward<T, AccT, VECSIZE>
    <<<grids, threads, dim * sizeof(AccT), stream>>>(
    out_data, in_data, N, dim);
}

template<typename T>
inline void LaunchD1BlockSharedSoftmaxForward(hipStream_t &stream, const T* in_data,
                        T* out_data, const int N, const int dim) {
  if(dim % 4 == 0) {
    LaunchD1BlockSharedSoftmaxForwardKernel<T, 4>(
      stream, in_data, out_data, N, dim);
  } else if(dim % 2 == 0) {
    LaunchD1BlockSharedSoftmaxForwardKernel<T, 2>(
      stream, in_data, out_data, N, dim);
  } else {
    LaunchD1BlockSharedSoftmaxForwardKernel<T, 1>(
      stream, in_data, out_data, N, dim);
  }
}

/*****************************************************************/
// when D == 1 && 4096 < dim, using KeD1BlockSoftmaxForward,
// each block compute a row, and synchronization by blockReduce,
// each thread compute VECSIZE elements of dim
template<typename T, typename AccT, int VECSIZE>
__global__ void KeD1BlockSoftmaxForward(T* __restrict__ dst,
            const T* __restrict__ src,const int N, const int dim) {
  const int tid = threadIdx.x;

  using VecT = typename GetVecType<T, VECSIZE>::type;
  VecT vec_src, vec_dst;// vectorization for global memory coalescing
  T* buf_src = reinterpret_cast<T*>(&vec_src);
  T* buf_dst = reinterpret_cast<T*>(&vec_dst);

  for(int row = blockIdx.x; row < N; row += gridDim.x) {
    const int offset = row * dim;
    const T* __restrict__ src_row = src + offset;
    T* __restrict__ dst_row = dst + offset;

    // compute max value
    AccT max_val = -std::numeric_limits<AccT>::infinity();
    for(int col = tid * VECSIZE; col < dim; col += blockDim.x * VECSIZE) {
      vec_src = reinterpret_cast<const VecT*>(&src_row[col])[0];
#pragma unroll
      for(int i = 0; i < VECSIZE; i ++) {
        max_val = max(static_cast<AccT>(buf_src[i]), max_val);
      }
    }
    max_val = math::blockReduceMax(max_val, 0xffffffff);
    // compute sum value
    AccT sum_val(0);
    for(int col = tid * VECSIZE; col < dim; col += blockDim.x * VECSIZE) {
      vec_src = reinterpret_cast<const VecT*>(&src_row[col])[0];
#pragma unroll
      for(int i = 0; i < VECSIZE; i ++) {
        sum_val += Exp(static_cast<AccT>(buf_src[i]) - max_val);
      }
    }
    sum_val = math::blockReduceSum(sum_val, 0xffffffff);
    // compute softmax result
    for(int col = tid * VECSIZE; col < dim; col += blockDim.x * VECSIZE) {
      vec_src = reinterpret_cast<const VecT*>(&src_row[col])[0];
#pragma unroll
      for(int i = 0; i < VECSIZE; i ++) {
        buf_dst[i] = static_cast<T>(
              Exp(static_cast<AccT>(buf_src[i]) - max_val) / (sum_val + 1e-6f));
      }
      reinterpret_cast<VecT*>(&dst_row[col])[0] = vec_dst;
    }
  }
}

template<typename T, int VECSIZE>
inline void LaunchD1BlockSoftmaxForwardKernel(hipStream_t &stream, const T* in_data,
                          T* out_data, const int N, const int dim) {
  const int threads = std::min(dim, 1024);
  const int grids = N;
  using AccT = typename GetAccType<T>::type;

  KeD1BlockSoftmaxForward<T, AccT, VECSIZE>
    <<<grids, threads, 0, stream>>>(
    out_data, in_data, N, dim);
}

template<typename T>
inline void LaunchD1BlockSoftmaxForward(hipStream_t &stream, const T* in_data,
                        T* out_data, const int N, const int dim) {
  if(dim % 4 == 0) {
    LaunchD1BlockSoftmaxForwardKernel<T, 4>(
      stream, in_data, out_data, N, dim);
  } else if(dim % 2 == 0) {
    LaunchD1BlockSoftmaxForwardKernel<T, 2>(
      stream, in_data, out_data, N, dim);
  } else {
    LaunchD1BlockSoftmaxForwardKernel<T, 1>(
      stream, in_data, out_data, N, dim);
  }
}

/*****************************************************************/
// When D is larg and dim is small:
// Each block arranged by N，each thread arranged by D
// each thread compute dim * VECSIZE number's softmax
template<typename T, typename AccT, int VECSIZE>
__global__ void KeLoopDimSoftmaxForward(T* __restrict__ dst,
      const T* __restrict__ src, const int N, const int dim, const int D) {
  assert(D % VECSIZE == 0);
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int vec_id = tid * VECSIZE;
  const int out_id = vec_id / D;
  if(out_id >= N) return;
  const int in_id = vec_id - out_id * D;
  // vectorization for global memory coalescing
  using VecT = typename GetVecType<T, VECSIZE>::type;
  VecT vec_src, vec_dst;
  T* buf_src = reinterpret_cast<T*>(&vec_src);
  T* buf_dst = reinterpret_cast<T*>(&vec_dst);

  const T* __restrict__ src_row = src + out_id * dim * D + in_id;
  T* __restrict__ dst_row = dst + out_id * dim * D + in_id;
  // compute max value
  AccT max_val[VECSIZE];
#pragma unroll
  for(int i = 0; i < VECSIZE; i ++) {
    max_val[i] = -std::numeric_limits<AccT>::infinity();
  }
  for(int dim_id = 0; dim_id < dim; dim_id ++) {
    vec_src = reinterpret_cast<const VecT*>(&src_row[dim_id * D])[0];
#pragma unroll
    for(int i = 0; i < VECSIZE; i ++) {
      max_val[i] = max(static_cast<AccT>(buf_src[i]), max_val[i]);
    }
  }
  // compute exponent value and sum value
  AccT sum_val[VECSIZE]{0};
  for(int dim_id = 0; dim_id < dim; dim_id ++) {
    vec_src = reinterpret_cast<const VecT*>(&src_row[dim_id * D])[0];
#pragma unroll
    for(int i = 0; i < VECSIZE; i ++) {
      sum_val[i] += Exp(static_cast<AccT>(buf_src[i]) - max_val[i]);
    }
  }

  // compute softmax value
  // TODO(jiangcheng): how to eliminate twice Exp
  for(int dim_id = 0; dim_id < dim; dim_id ++) {
    vec_src = reinterpret_cast<const VecT*>(&src_row[dim_id * D])[0];
#pragma unroll
    for(int i = 0; i < VECSIZE; i ++) {
      buf_dst[i] = static_cast<T>(
          Exp(static_cast<AccT>(buf_src[i]) - max_val[i]) /
          (sum_val[i] + 1e-6f));
    }
    reinterpret_cast<VecT*>(&dst_row[dim_id * D])[0] = vec_dst;
  }
}

template<typename T, int VECSIZE>
inline void LaunchLoopDimSoftmaxForwardKernel(hipStream_t &stream, const T* in_data,
                  T* out_data, const int N, const int dim, const int D) {
  int loop_num = N * D / VECSIZE;
  int threads = std::min(loop_num, 1024);
  int grids = (loop_num + threads - 1) / threads;
  using AccT = typename GetAccType<T>::type;

  KeLoopDimSoftmaxForward<T, AccT, VECSIZE>
      <<<grids, threads, 0, stream>>>(
      out_data, in_data, N, dim, D);
}

template<typename T>
inline void LaunchLoopDimSoftmaxForward(hipStream_t &stream, const T* in_data,
                  T* out_data, const int N, const int dim, const int D) {
  if(D % 4 == 0) {
    LaunchLoopDimSoftmaxForwardKernel<T, 4>(
      stream, in_data, out_data, N, dim, D);
  } else if (D % 2 == 0) {
    LaunchLoopDimSoftmaxForwardKernel<T, 2>(
      stream, in_data, out_data, N, dim, D);
  } else {
    LaunchLoopDimSoftmaxForwardKernel<T, 1>(
      stream, in_data, out_data, N, dim, D);
  }
}

/*****************************************************************/

// When D is small and (dim * D) is larger
// Each block arranged by N，each thread arranged by dim * D
// each block compute (dim * D) number's softmax
template<typename T, typename AccT, int VECSIZE>
__global__ void KeSpandDimDSoftmaxForward(T* __restrict__ dst,
      const T* __restrict__ src, const int N, const int dim, const int D) {
  extern __shared__ __align__(sizeof(AccT)) unsigned char s_mem[];
  AccT* s_data = reinterpret_cast<AccT*>(s_mem);
  // vectorization for global memory coalescing
  using VecT = typename GetVecType<T, VECSIZE>::type;
  VecT vec_src, vec_dst;
  T* buf_src = reinterpret_cast<T*>(&vec_src);
  T* buf_dst = reinterpret_cast<T*>(&vec_dst);

  const int tid = threadIdx.x;
  const int vec_id = tid * VECSIZE;
  const int BlockDim = blockDim.x;
  const int vec_num = BlockDim * VECSIZE;
  for(int out_id = blockIdx.x; out_id < N; out_id += gridDim.x) {
    const T* __restrict__ src_row = src + out_id * dim * D;
    T* __restrict__ dst_row = dst + out_id * dim * D;

    // Compute each thread's max value
    AccT max_val[VECSIZE];
#pragma unroll
    for(int i = 0; i < VECSIZE; i ++) {
      max_val[i] = -std::numeric_limits<AccT>::infinity();
    }
    for(int id = vec_id; id < dim * D; id += vec_num) {
      vec_src = reinterpret_cast<const VecT*>(&src_row[id])[0];
#pragma unroll
      for(int i = 0; i < VECSIZE; i ++) {
        max_val[i] = max(static_cast<AccT>(buf_src[i]), max_val[i]);
      }
    }
    // write to shared memory
#pragma unroll
    for(int i = 0; i < VECSIZE; i ++) {
      s_data[vec_id + i] = max_val[i];
    }
    __syncthreads();
    // compute total max value
#pragma unroll
    for(int i = 0; i < VECSIZE; i ++) {
      for(int k = (vec_id + i) % D; k < vec_num; k += D) {
        max_val[i] = max(s_data[k], max_val[i]);
      }
    }
    // Compute each thread's sum value
    AccT sum_val[VECSIZE]{0};
    for(int id = vec_id; id < dim * D; id += vec_num) {
      vec_src = reinterpret_cast<const VecT*>(&src_row[id])[0];
#pragma unroll
      for(int i = 0; i < VECSIZE; i ++) {
        sum_val[i] += Exp(static_cast<AccT>(buf_src[i]) - max_val[i]);
      }
    }
    // write to shared memory
#pragma unroll
    for(int i = 0; i < VECSIZE; i ++) {
      s_data[vec_id + i] = sum_val[i];
    }
    __syncthreads();
    // compute total sum value
#pragma unroll
    for(int i = 0; i < VECSIZE; i ++) {
      sum_val[i] = 0;
      for(int k = (vec_id + i) % D; k < vec_num; k += D) {
        sum_val[i] += s_data[k];
      }
    }
    // Compute finally softmax result
    // TODO(jiangcheng): how to eliminate twice Exp
    for(int id = vec_id; id < dim * D; id += vec_num) {
      vec_src = reinterpret_cast<const VecT*>(&src_row[id])[0];
#pragma unroll
      for(int i = 0; i < VECSIZE; i ++) {
        buf_dst[i] = static_cast<T>(
          Exp(static_cast<AccT>(buf_src[i]) - max_val[i]) /
          (sum_val[i] + 1e-6f));
      }
      reinterpret_cast<VecT*>(&dst_row[id])[0] = vec_dst;
    }
  }
}

template<typename T, int VECSIZE>
inline void LaunchSpandDimDSoftmaxForwardKernel(hipStream_t &stream,
                        const T* in_data, T* out_data,
                        const int N, const int dim, const int D) {
  const int grids = N;
  const int threads = std::min(dim * D, 256);
  using AccT = typename GetAccType<T>::type;

  KeSpandDimDSoftmaxForward<T, AccT, VECSIZE>
    <<<grids, threads, threads * VECSIZE * sizeof(AccT), stream>>>(
    out_data, in_data, N, dim, D);
}

template<typename T>
inline void LaunchSpandDimDSoftmaxForward(hipStream_t &stream,
                        const T* in_data, T* out_data,
                        const int N, const int dim, const int D) {
  const int cols = dim * D;
  if(cols % 4 == 0) {
    LaunchSpandDimDSoftmaxForwardKernel<T, 4>(
      stream, in_data, out_data, N, dim, D);
  } else if(cols % 2 == 0) {
    LaunchSpandDimDSoftmaxForwardKernel<T, 2>(
      stream, in_data, out_data, N, dim, D);
  } else {
    LaunchSpandDimDSoftmaxForwardKernel<T, 1>(
      stream, in_data, out_data, N, dim, D);
  }
}

template <typename T, typename AccT, int Log2Elements>
__global__ void softmax_warp_backward(T* gradInput, const T* grad,
                                      const T* output, int batch_size,
                                      int stride, int element_count) {
  constexpr int next_power_of_two = 1 << Log2Elements;
  constexpr int warp_size_softmax =
      (next_power_of_two < 32) ? next_power_of_two : 32;
  constexpr int WARP_ITERATIONS = next_power_of_two / warp_size_softmax;
  constexpr int WARP_BATCH = (next_power_of_two <= 128) ? 2 : 1;

  int first_batch = (blockDim.y * blockIdx.x + threadIdx.y) * WARP_BATCH;

  int local_batches = batch_size - first_batch;
  if (local_batches > WARP_BATCH) {
    local_batches = WARP_BATCH;
  }

  int local_idx = threadIdx.x % warp_size_softmax;

  int thread_offset = first_batch * stride + local_idx;
  grad += thread_offset;
  output += thread_offset;
  gradInput += thread_offset;

  // load data from global memory
  AccT grad_reg[WARP_BATCH][WARP_ITERATIONS];
  AccT output_reg[WARP_BATCH][WARP_ITERATIONS];
  for (int i = 0; i < WARP_BATCH; ++i) {
    int batch_element_count = (i >= local_batches) ? 0 : element_count;
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      int element_index = local_idx + it * warp_size_softmax;
      if (element_index < batch_element_count) {
        grad_reg[i][it] =
            static_cast<AccT>(grad[i * element_count + it * warp_size_softmax]);
        output_reg[i][it] = static_cast<AccT>(
            output[i * element_count + it * warp_size_softmax]);
      } else {
        grad_reg[i][it] = AccT(0);
        output_reg[i][it] = AccT(0);
      }
    }
  }

  AccT sum[WARP_BATCH];
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
    sum[i] = grad_reg[i][0];
#pragma unroll
    for (int it = 1; it < WARP_ITERATIONS; ++it) {
      sum[i] += grad_reg[i][it];
    }
  }
  warp_reduce_sum<AccT, WARP_BATCH, warp_size_softmax>(sum);

// store result
#pragma unroll
  for (int i = 0; i < WARP_BATCH; ++i) {
    if (i >= local_batches) break;
#pragma unroll
    for (int it = 0; it < WARP_ITERATIONS; ++it) {
      int element_index = local_idx + it * warp_size_softmax;
      if (element_index < element_count) {
        // compute gradients
        gradInput[i * element_count + it * warp_size_softmax] =
            (grad_reg[i][it] - output_reg[i][it] * sum[i]);
      }
    }
  }
}

// When D is small and (dim * D) is larger
// Each block arranged by N，each thread arranged by dim * D
// each block compute (dim * D) number's softmax
template<typename T, typename AccT>
__global__ void KeSpandDimDSoftmaxBackward(T *dx, const T *out, const T *dout,
                        const int N, const int dim, const int D) {
  extern __shared__ __align__(sizeof(AccT)) unsigned char s_mem[];
  AccT* s_data = reinterpret_cast<AccT*>(s_mem);

  const int tid = threadIdx.x;
  const int BlockDim = blockDim.x;
  for(int out_id = blockIdx.x; out_id < N; out_id += gridDim.x) {
    const T *src_out = out + out_id * dim * D;
    const T *src_dout = dout + out_id * dim * D;
    T *dst_dx = dx + out_id * dim * D;

    // Compute each thread's sum value
    AccT sum_val(0);
    for(int id = tid; id < dim * D; id += BlockDim)
      sum_val += static_cast<AccT>(src_out[id]) *
                 static_cast<AccT>(src_dout[id]);
    // write to shared memory
    s_data[tid] = sum_val;
    __syncthreads();
    // compute total sum value
    sum_val = 0;
    for(int id = tid % D; id < BlockDim; id += D)
      sum_val += s_data[id];

    // Compute finally softmax result
    for(int id = tid; id < dim * D; id += BlockDim)
      dst_dx[id] =
        static_cast<T>(static_cast<AccT>(src_out[id]) *
        (static_cast<AccT>(src_dout[id]) - sum_val));
  }
}

template <typename T>
class SoftmaxCUDNNKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* x = ctx.Input<Tensor>("X");
    auto* out = ctx.Output<Tensor>("Out");
    out->mutable_data<T>(ctx.GetPlace());
    auto* out_data = out->data<T>();

    auto dims = x->dims();
    const int rank = dims.size();
    const int axis = CanonicalAxis(ctx.Attr<int>("axis"), rank);
    const int dim = dims[axis];
    const int N = SizeToAxis(axis, dims);
    const int D = SizeOutAxis(axis, dims);

    constexpr int max_dim = 320;
    bool optimize = false;
    auto stream = ctx.cuda_device_context().stream();
    if (D == 1) {
      if (dim < max_dim && sizeof(T) <= 4) {
        optimize = true;
        LaunchWarpSoftmaxForward(stream, x->data<T>(), out_data, N, dim);
      } else if(dim <= 1024) {
        optimize = true;
        LaunchD1WarpSoftmaxForward(stream, x->data<T>(), out_data, N, dim);
      } else if(dim <= 4096) {
        optimize = true;
        LaunchD1BlockSharedSoftmaxForward(stream, x->data<T>(), out_data, N, dim);
      } else {
        optimize = true;
        LaunchD1BlockSoftmaxForward(stream, x->data<T>(), out_data, N, dim);
      }
    } else {
      if(D <= 256) {
        optimize = true;
        LaunchSpandDimDSoftmaxForward(stream, x->data<T>(), out_data, N, dim, D);
      } else if(dim <= 512) {
        optimize = true;
        LaunchLoopDimSoftmaxForward(stream, x->data<T>(), out_data, N, dim, D);
      }
    }
    if (!optimize) {
      ScopedTensorDescriptor desc;
      std::vector<int> tensor_dims = {N, dim, D, 1};
      DataLayout layout = DataLayout::kNCHW;
#ifdef PADDLE_WITH_HIP
      miopenTensorDescriptor_t desc_ = desc.descriptor<T>(layout, tensor_dims);
#else
      hipdnnTensorDescriptor_t desc_ = desc.descriptor<T>(layout, tensor_dims);
#endif

      auto& dev_ctx =
          ctx.template device_context<platform::CUDADeviceContext>();
      auto handle = dev_ctx.cudnn_handle();

#ifdef PADDLE_WITH_HIP
      auto mode = axis == rank - 1 ? MIOPEN_SOFTMAX_MODE_INSTANCE
                                   : MIOPEN_SOFTMAX_MODE_CHANNEL;
      PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::miopenSoftmaxForward(
          handle, platform::CudnnDataType<T>::kOne(), desc_, x->data<T>(),
          platform::CudnnDataType<T>::kZero(), desc_, out_data));
#else
      auto mode = axis == rank - 1 ? HIPDNN_SOFTMAX_MODE_INSTANCE
                                   : HIPDNN_SOFTMAX_MODE_CHANNEL;
      PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSoftmaxForward(
          handle, HIPDNN_SOFTMAX_ACCURATE, mode,
          platform::CudnnDataType<T>::kOne(), desc_, x->data<T>(),
          platform::CudnnDataType<T>::kZero(), desc_, out_data));
#endif
    }
  }
};

template <typename T>
class SoftmaxGradCUDNNKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto* out = ctx.Input<Tensor>("Out");
    auto* dout = ctx.Input<Tensor>(framework::GradVarName("Out"));
    auto* dx = ctx.Output<Tensor>(framework::GradVarName("X"));
    dx->mutable_data<T>(ctx.GetPlace());
    auto* dx_data = dx->data<T>();

    auto dims = out->dims();
    const int rank = dims.size();
    const int axis = CanonicalAxis(ctx.Attr<int>("axis"), rank);
    const int dim = dims[axis];
    const int N = SizeToAxis(axis, dims);
    const int D = SizeOutAxis(axis, dims);

    constexpr bool AccT_use_float =
        std::is_same<T, float>::value ||
        std::is_same<T, platform::float16>::value;
    bool optimize = false;
    if(D <= 1024) {
      optimize = true;
      const int grids = N;
      const int threads = D * (1024 / D);

      if(AccT_use_float) {
        KeSpandDimDSoftmaxBackward<T, float>
          <<<grids, threads, threads * sizeof(float),
          ctx.cuda_device_context().stream()>>>(
          dx_data, out->data<T>(), dout->data<T>(), N, dim, D);
      } else {
        KeSpandDimDSoftmaxBackward<T, double>
          <<<grids, threads, threads * sizeof(double),
          ctx.cuda_device_context().stream()>>>(
          dx_data, out->data<T>(), dout->data<T>(), N, dim, D);
      }
    }
    if (!optimize) {
      ScopedTensorDescriptor desc;
      std::vector<int> tensor_dims = {N, dim, D, 1};
      DataLayout layout = DataLayout::kNCHW;
#ifdef PADDLE_WITH_HIP
      miopenTensorDescriptor_t desc_ = desc.descriptor<T>(layout, tensor_dims);
#else
      hipdnnTensorDescriptor_t desc_ = desc.descriptor<T>(layout, tensor_dims);
#endif

      auto& dev_ctx =
          ctx.template device_context<platform::CUDADeviceContext>();
      auto handle = dev_ctx.cudnn_handle();

#ifdef PADDLE_WITH_HIP
      auto mode = axis == rank - 1 ? MIOPEN_SOFTMAX_MODE_INSTANCE
                                   : MIOPEN_SOFTMAX_MODE_CHANNEL;
      PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::miopenSoftmaxBackward(
          handle, platform::CudnnDataType<T>::kOne(), desc_, out->data<T>(),
          desc_, dout->data<T>(), platform::CudnnDataType<T>::kZero(), desc_,
          dx_data));
#else
      auto mode = axis == rank - 1 ? HIPDNN_SOFTMAX_MODE_INSTANCE
                                   : HIPDNN_SOFTMAX_MODE_CHANNEL;
      PADDLE_ENFORCE_CUDA_SUCCESS(platform::dynload::hipdnnSoftmaxBackward(
          handle, HIPDNN_SOFTMAX_ACCURATE, mode,
          platform::CudnnDataType<T>::kOne(), desc_, out->data<T>(), desc_,
          dout->data<T>(), platform::CudnnDataType<T>::kZero(), desc_,
          dx_data));
#endif
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
#ifdef PADDLE_WITH_HIP
// MIOPEN do not support double
REGISTER_OP_KERNEL(softmax, CUDNN, plat::CUDAPlace,
                   ops::SoftmaxCUDNNKernel<float>,
                   ops::SoftmaxCUDNNKernel<plat::float16>);
REGISTER_OP_KERNEL(softmax_grad, CUDNN, plat::CUDAPlace,
                   ops::SoftmaxGradCUDNNKernel<float>,
                   ops::SoftmaxGradCUDNNKernel<plat::float16>);
#else
REGISTER_OP_KERNEL(softmax, CUDNN, plat::CUDAPlace,
                   ops::SoftmaxCUDNNKernel<float>,
                   ops::SoftmaxCUDNNKernel<double>,
                   ops::SoftmaxCUDNNKernel<plat::float16>);
REGISTER_OP_KERNEL(softmax_grad, CUDNN, plat::CUDAPlace,
                   ops::SoftmaxGradCUDNNKernel<float>,
                   ops::SoftmaxGradCUDNNKernel<double>,
                   ops::SoftmaxGradCUDNNKernel<plat::float16>);
#endif
