#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <fstream>

#include "stdio.h"
#include "time.h"

template <typename T, int BlockDimX, int BlockDimY, int GridDimX>
__global__ void LookupTableV2Grad(T *table, const T *output, const int64_t *ids,
                                  const int64_t N, const int64_t K,
                                  const int64_t D) {
  int idx = threadIdx.x;
  int idy = blockIdx.x + threadIdx.y * GridDimX;

  while (idy < K) {
    int64_t id = ids[idy];
    const T *out = output + idy * D;
    T *tab = table + id * D;
    for (int i = idx; i < D; i += BlockDimX) {
      atomicAdd(&tab[i], out[i]);
    }
    idy += BlockDimY * GridDimX;
  }
}

template <typename T, int Tile>
__global__ void LookupTableV2Grad2(T *table, const T *output, const int64_t *ids,
                                  const int64_t N, const int64_t K,
                                  const int64_t D) {
  /*
    data:    out0 out1 ...  out7  out0   out1 ... out7
    row:     ids0 ids0 ...  ids0  ids1   ids1 ... ids3  
      |       |     |         |     |     |         |
      V       V     V         V     V     V         V
    tile:   tile0 tile1 ... tile7 tile0 tile1 ... tile7
      |       |     |         |     |     |         |
      V       V     V         V     V     V         V
    thread:   t0   t1   ...  t7    t8    t9   ...  t31  
    
    each tile (4 threads) atomicAdd K data
  */
                          
  const int tid = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y;
  const int warp_id = tid / 32, tid_of_warp = tid % 32;
  const int tile_num = (gridDim.x * blockDim.x * blockDim.y + Tile - 1) / Tile;
  const int tile_of_warp = 32 / Tile;  // tile_of_warp = 8
  int tile_id = warp_id * tile_of_warp + tid_of_warp % 8;

  while(tile_id < D) {
    for(int i = tid_of_warp / tile_of_warp; i < K; i += Tile) {
      int64_t id = ids[i];
      const T *out = output + i * D;
      T *tab = table + id * D;

      atomicAdd(&tab[tile_id], out[tile_id]);
    }
    tile_id += tile_num;
  }
}

template<typename T>
float Convert2Float(T input) {
    return static_cast<float>(input);
}

template<>
float Convert2Float<half>(half input) {
    return __half2float(input);
}
template<typename T>
T* MallocDevice(int64_t size) {
    T *ptr_d;
    hipMalloc((void**)&ptr_d, size * sizeof(T));
    return ptr_d;
}
template<typename T>
T* MallocHost(int64_t size) {
    T *ptr_h;
    hipHostMalloc((void**)&ptr_h, size * sizeof(T), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);
    return ptr_h;
}
template<typename T>
void SetZero(T *ptr, int64_t size, hipStream_t &context) {
    hipMemsetAsync(ptr, 0, size * sizeof(T), context);
}

template<typename T>
void Host2Device(T *des, T *src, int64_t size, hipStream_t &context) {
    hipMemcpyAsync(des, src, size * sizeof(T), hipMemcpyHostToDevice, context);
}
template<typename T>
void Device2Host(T *des, T *src, int64_t size, hipStream_t &context) {
    hipMemcpyAsync(des, src, size * sizeof(T), hipMemcpyDeviceToHost, context);
}

template<typename T>
double MaxError(T *data_1, T *data_2, int64_t size) {
    double maxerr = 0.0, err = 0.0;
    for(int i = 0; i < size; i ++) {
        err = fabs(Convert2Float<T>(data_1[i]) - Convert2Float<T>(data_2[i]));
        if(err > maxerr) maxerr = err;
    }
    return maxerr;
}
template<typename T>
void Print2File(T *data, int64_t N, int64_t D, const char *filename) {
    std::fstream file(filename, std::ios_base::out);
    for(int i = 0; i < N; i ++) {
        for(int j = 0; j < D; j ++) {
            file<<Convert2Float<T>(data[i * D + j])<< " ";
        }
        file<<std::endl;
    }
}

void TestKernel(int N, int K, int D, hipStream_t &context) {
    dim3 threads(128, 8);
    dim3 grids(8, 1);
    
    //float
    float *table_d32_1 = MallocDevice<float>(N * D), *table_d32_2 = MallocDevice<float>(N * D);
    int64_t *ids_h = MallocHost<int64_t>(K), *ids_d = MallocDevice<int64_t>(K);
    float *out_h_32 = MallocHost<float>(K * D), *out_d_32 = MallocDevice<float>(K * D);

    SetZero(table_d32_1, N * D, context);
    SetZero(table_d32_2, N * D, context);
    for(int i = 0; i < K; i ++) {
        ids_h[i] = rand() % N;
    }
    Host2Device(ids_d, ids_h, K, context);
    for(int i = 0; i < K * D; i ++) {
         out_h_32[i] = (rand() % 2000000 - 1000000.0f) / 100000.0f;
    }
    Host2Device(out_d_32, out_h_32, K * D, context);

    LookupTableV2Grad<float, 128, 8, 8><<<grids, threads, 0, context>>>(
          table_d32_1, out_d_32, ids_d, N, K, D);
    LookupTableV2Grad2<float, 4><<<grids, threads, 0, context>>>(
          table_d32_2, out_d_32, ids_d, N, K, D);

    float *table_h32_1 = MallocHost<float>(N * D), *table_h32_2 = MallocHost<float>(N * D);
    Device2Host(table_h32_1, table_d32_1, N * D, context);
    Device2Host(table_h32_2, table_d32_2, N * D, context);
    
    //half
    half *table_d16_1 = MallocDevice<half>(N * D), *table_d16_2 = MallocDevice<half>(N * D);
    half *out_h_16 = MallocHost<half>(K * D), *out_d_16 = MallocDevice<half>(K * D);

    SetZero(table_d16_1, N * D, context);
    SetZero(table_d16_2, N * D, context);
    for(int i = 0; i < K * D; i ++) {
        out_h_16[i] = __float2half(out_h_32[i]);
    }
    Host2Device(out_d_16, out_h_16, K * D, context);

    LookupTableV2Grad<half, 128, 8, 8><<<grids, threads, 0, context>>>(
          table_d16_1, out_d_16, ids_d, N, K, D);
    LookupTableV2Grad2<half, 4><<<grids, threads, 0, context>>>(
          table_d16_2, out_d_16, ids_d, N, K, D);

    half *table_h16_1 = MallocHost<half>(N * D), *table_h16_2 = MallocHost<half>(N * D);
    Device2Host(table_h16_1, table_d16_1, N * D, context);
    Device2Host(table_h16_2, table_d16_2, N * D, context);

    //check result
    hipStreamSynchronize(context);
    double maxerr32 = MaxError(table_h32_1, table_h32_2, N * D);
    double maxerr16 = MaxError(table_h16_1, table_h16_2, N * D);

    printf("fp32 maxerr %f and fp16 maxerr %f\n", maxerr32, maxerr16);

    // out to file
    Print2File(table_h32_1, N, D, "table_fp32_1.log");
    Print2File(table_h32_2, N, D, "table_fp32_2.log");
    Print2File(table_h16_1, N, D, "table_fp16_1.log");
    Print2File(table_h16_2, N, D, "table_fp16_2.log");

    // free device memory
    hipFree(table_d32_1);
    hipFree(table_d32_2);
    hipFree(table_d16_1);
    hipFree(table_d16_2);
    hipFree(ids_d);
    hipFree(out_d_32);
    hipFree(out_d_16);

    // free host memory
    hipHostFree(table_h32_1);
    hipHostFree(table_h32_2);
    hipHostFree(table_h16_1);
    hipHostFree(table_h16_2);
    hipHostFree(ids_h);
    hipHostFree(out_h_32);
    hipHostFree(out_h_16);
}

int main() {
    int64_t N, K, D;
    N = K = D = 1024;

    srand((unsigned)time(NULL));
    hipStream_t context;
    hipStreamCreate(&context);

    TestKernel(N, K, D, context);

    hipStreamDestroy(context);
    return 0;
}