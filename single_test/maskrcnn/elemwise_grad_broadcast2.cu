#include "hip/hip_runtime.h"
#include "../common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

template <typename T, typename DX_OP, typename DY_OP>
static __global__ void ElemwiseGradBroadcast2CUDAKernel(
    const T *x, const T *y, const T *out, const T *dout, int pre, int n,
    int post, bool is_xsize_larger, DX_OP dx_op, DY_OP dy_op, T *dx, T *dy) {
  int tid = threadIdx.x;
  int j = blockIdx.x;

  T val(0);
  int ttid = tid;

  if (is_xsize_larger) {
    while (true) {
      int i = ttid / post;
      int k = ttid % post;
      if (i >= pre) break;

      int x_offset = i * n * post + j * post + k;

      if (dx != nullptr) {
        dx[x_offset] = dx_op(x[x_offset], y[j], out[x_offset], dout[x_offset]);
      }

      if (dy != nullptr) {
        val += dy_op(x[x_offset], y[j], out[x_offset], dout[x_offset]);
      }

      ttid += ELEMWISE_MAX_BLOCK_DIM;
    }

    if (dy) {
      int h = pre * post;
      h = h > ELEMWISE_MAX_BLOCK_DIM ? ELEMWISE_MAX_BLOCK_DIM : h;
      val = paddle::platform::reduceSum(val, tid, h);
      if (threadIdx.x == 0) {
        dy[j] = val;
      }
    }
  } else {  // x.dims < y.dims, broadcast for x.
    while (true) {
      int i = ttid / post;
      int k = ttid % post;
      if (i >= pre) break;

      int y_offset = i * n * post + j * post + k;

      if (dy != nullptr) {
        dy[y_offset] = dy_op(x[j], y[y_offset], out[y_offset], dout[y_offset]);
      }

      if (dx != nullptr) {
        val += dx_op(x[j], y[y_offset], out[y_offset], dout[y_offset]);
      }

      ttid += ELEMWISE_MAX_BLOCK_DIM;
    }

    if (dx) {
      int h = pre * post;
      h = h > ELEMWISE_MAX_BLOCK_DIM ? ELEMWISE_MAX_BLOCK_DIM : h;
      val = paddle::platform::reduceSum(val, tid, h);
      if (threadIdx.x == 0) {
        dx[j] = val;
      }
    }
  }
}

template <typename T, typename DX_OP, typename DY_OP>
static void ElemwiseGradBroadcast2CUDA(hipStream_t stream, const T *x,
                                       const T *y, const T *out, const T *dout,
                                       int pre, int n, int post,
                                       bool is_xsize_larger, DX_OP dx_op,
                                       DY_OP dy_op, T *dx, T *dy) {
  int block_size = std::min(ELEMWISE_MAX_BLOCK_DIM, pre * post);
  int gird_size = n;
  ElemwiseGradBroadcast2CUDAKernel<<<gird_size, block_size, 0, stream>>>(
      x, y, out, dout, pre, n, post, is_xsize_larger, dx_op, dy_op, dx, dy);
}