#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "stdio.h"
#include "time.h"

template<typename T>
__global__ void KeAtomic(T *data, int size, T *out) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size) {
        atomicAdd(&out[idx], data[idx]);
    }
}

template<typename T>
float convert2float(T input) {
    return static_cast<float>(input);
}

template<>
float convert2float<half>(half input) {
    return __half2float(input);
}

template<typename TYPE>
float TimeOfKernel(int size, hipStream_t &context) {
    TYPE *table_h, *table_d;
    hipHostMalloc((void**)&table_h, size * sizeof(TYPE), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);
    hipMalloc((void**)&table_d, size * sizeof(TYPE));

    for(int i = 0; i < size; i ++) {
        table_h[i] = 0.00001f * i;
    }
    hipMemcpyAsync(table_d, table_h, size * sizeof(TYPE), hipMemcpyHostToDevice, context);

    TYPE *output_h, *output_d;
    hipHostMalloc(&output_h, size * sizeof(TYPE), hipHostMallocDefault, hipHostMallocDefault);
    hipMalloc(&output_d, size * sizeof(TYPE));
    hipMemset(output_d, 0, size * sizeof(TYPE));

    int threads = 1024;
    int grids = (size + threads - 1) / threads;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, context);
    KeAtomic<TYPE><<<grids, threads, 0, context>>>(
          table_d, size, output_d);
    hipEventRecord(stop, context);
    hipEventSynchronize(stop);

    float time_of_kernel;
    hipEventElapsedTime(&time_of_kernel, start, stop);

#if 0
    cudaMemcpyAsync(output_h, output_d, size * sizeof(TYPE), cudaMemcpyDeviceToHost, context);
    cudaStreamSynchronize(context);
    for(int i = 0; i < 10; i++) {
        printf("%f ", convert2float<TYPE>(output_h[i]));
    }
    printf("\n");
#endif

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(table_d);
    hipHostFree(table_h);
    hipFree(output_d);
    hipHostFree(output_h);

    return time_of_kernel;
}

int main() {
    srand((unsigned)time(NULL));
    hipStream_t context;
    hipStreamCreate(&context);

    
    for(int i = 1024 * 1024; i <= 1024 * 1024; i <<= 1) {
        float t_fp13 = TimeOfKernel<float>(i, context);
        float t_fp16 = TimeOfKernel<half>(i, context);
        printf("atomic %d time fp32 %f ms vs fp16 %f ms\n", i, t_fp13, t_fp16);
    }

    hipStreamDestroy(context);
    return 0;
}